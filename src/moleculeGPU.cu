#include "molecule.cuh"
#include "kernels.cuh"

void MoleculeGPU::coefTensor()
{
    atomCoofTensor << <1, countThread, 0, CudaStream >> > (d_ABCDEF, d_xA, d_yA, d_zA, M);
    hipMemcpyAsync(ABCDEF, d_ABCDEF, 6 * sizeof(float), hipMemcpyDeviceToHost, CudaStream);
    hipDeviceSynchronize();
    //std::cout <<"A: " << h_ABCDEF[0] << ", B: " << h_ABCDEF[1] << ", C: " << h_ABCDEF[2] << ", D: " << h_ABCDEF[3] << ", E: " << h_ABCDEF[4] << ", F: " << h_ABCDEF[5] << std::endl;
}
MoleculeGPU::MoleculeGPU(std::string name, std::string nameStructure, int N, int n) : Molecule(name, nameStructure, N, n)
{
    err = hipStreamCreate(&CudaStream);
    countThread = (M / 32 + 1) * 32;
    //Device allocate
    err = hipMalloc((void**)&d_xA, sizeM); err = hipMalloc((void**)&d_yA, sizeM); err = hipMalloc((void**)&d_zA, sizeM);
    err = hipMalloc((void**)&d_uA, sizeM); err = hipMalloc((void**)&d_vA, sizeM); err = hipMalloc((void**)&d_wA, sizeM);
    err = hipMalloc((void**)&d_xA0, sizeM); err = hipMalloc((void**)&d_yA0, sizeM); err = hipMalloc((void**)&d_zA0, sizeM);
    err = hipMalloc((void**)&d_xA1, sizeM); err = hipMalloc((void**)&d_yA1, sizeM); err = hipMalloc((void**)&d_zA1, sizeM);
    err = hipMalloc((void**)&d_xFLJ, sizeM); err = hipMalloc((void**)&d_yFLJ, sizeM); err = hipMalloc((void**)&d_zFLJ, sizeM);

    //Host to device copy
    err = hipMemcpyAsync(d_xA, xA, sizeM, hipMemcpyHostToDevice, CudaStream);
    err = hipMemcpyAsync(d_yA, yA, sizeM, hipMemcpyHostToDevice, CudaStream);
    err = hipMemcpyAsync(d_zA, zA, sizeM, hipMemcpyHostToDevice, CudaStream);
    //OutputData
    err = hipHostAlloc((void**)&ABCDEF, 6 * sizeof(float), hipHostMallocDefault);
    err = hipMalloc((void**)&d_ABCDEF, 6 * sizeof(float));
    err = hipHostAlloc((void**)&UVW, 3 * sizeof(float), hipHostMallocDefault);
    err = hipMalloc((void**)&d_UVW, 3 * sizeof(float));
    err = hipHostAlloc((void**)&Lxyz, 3 * sizeof(float), hipHostMallocDefault);
    err = hipMalloc((void**)&d_Lxyz, 3 * sizeof(float));
    hipDeviceSynchronize();
}
MoleculeGPU::~MoleculeGPU()
{
    hipStreamDestroy(CudaStream);
    hipFree(d_xA); hipFree(d_yA); hipFree(d_zA);
    hipFree(d_uA); hipFree(d_vA); hipFree(d_wA);
    hipFree(d_xA0); hipFree(d_yA0); hipFree(d_zA0);
    hipFree(d_xA1); hipFree(d_yA1); hipFree(d_zA1);
    hipFree(d_xFLJ); hipFree(d_yFLJ); hipFree(d_zFLJ);
    hipHostFree(ABCDEF); hipFree(d_ABCDEF);
    hipHostFree(UVW); hipFree(d_UVW);
    hipHostFree(Lxyz); hipFree(d_Lxyz);
}
void MoleculeGPU::prepareStep()
{
    Molecule::prepareStep();
    atomPrepareStepD << <1, countThread, 0, CudaStream >> > (d_xA, d_yA, d_zA, d_xA0, d_yA0, d_zA0, d_xA1, d_yA1, d_zA1, xC, yC, zC, xM, yM, zM, M, n);
    hipDeviceSynchronize();
}
void MoleculeGPU::RungeKuttaStep(float dt)
{
    Molecule::RungeKuttaStep(dt);
    atomVelocity << <1, countThread, 0, CudaStream >> > (d_xA, d_yA, d_zA, d_xA1, d_yA1, d_zA1, d_uA, d_vA, d_wA, omegaX, omegaY, omegaZ, M, dt);
    hipDeviceSynchronize();
}
void MoleculeGPU::moleculeStep(float dt)
{
    Molecule::moleculeStep(dt);
    atomStep << <1, countThread, 0, CudaStream >> > (d_xA, d_yA, d_zA, d_xA0, d_yA0, d_zA0, xC, yC, zC, d_uA, d_vA, d_wA, xM, yM, zM, n, M, dt);
    hipDeviceSynchronize();
    coefTensor();
    angleVelocity();
}
void MoleculeGPU::calculateFLJ() //+velocity Step
{
    atomCalculateFLJ << <1, countThread, countThread * sizeof(float), CudaStream >> > (xC, yC, zC, d_UVW, d_Lxyz, N, M, n);  //���� ��� ������� ��� ������ � ��������� ��������� sm;
    err = hipMemcpyAsync(UVW, d_UVW, 3 * sizeof(float), hipMemcpyDeviceToHost, CudaStream);
    hipDeviceSynchronize();
    ub = UVW[0] / massM; vb = UVW[1] / massM; wb = UVW[2] / massM;
    //std::cout << "ub: " << ub << ", vb: " << vb << ", wb: " << wb << std::endl;
    err = hipMemcpyAsync(Lxyz, d_Lxyz, 3 * sizeof(float), hipMemcpyDeviceToHost, CudaStream);
    hipDeviceSynchronize();
    Lx = Lxyz[0]; Ly = Lxyz[1]; Lz = Lxyz[2];
    //std::cout << "Lx: " << Lx << ", Ly: " << Ly << ", Lz: " << Lz << std::endl;
}